#include "hip/hip_runtime.h"
#pragma once

#include "LongDouble.cuh"



using namespace std;

void LongDoubleCPU::initFromString(const string& s) {
	size_t index;

	if (s[0] == '-') {
		sign = -1;
		index = 1;
	}
	else {
		sign = 1;
		index = 0;
	}

	exponent = s.length() - index;

	while (index < s.length()) {
		if (s[index] == '.')
			exponent = sign == 1 ? index : index - 1;
		else
			digits.push_back(s[index] - '0');

		index++;
	}
}

void LongDoubleCPU::removeZeroes() {
	size_t n = max((long)1, exponent);

	while (digits.size() > n && digits[digits.size() - 1] == 0)
		digits.erase(digits.end() - 1);

	while (digits.size() > 1 && digits[0] == 0) {
		digits.erase(digits.begin());
		exponent--;
	}

	while (digits.size() > 1 && digits[digits.size() - 1] == 0)
		digits.erase(digits.end() - 1);

	if (isZero()) {
		exponent = 1;
		sign = 1;
	}

	normalize();
}

void LongDoubleCPU::normalize() {
	size_t start = max(exponent, (long)0);
	size_t realDigits = digits.size() - start;

	if (realDigits >= divDigits) {
		size_t count = 0;
		size_t maxCount = 0;

		size_t i = start;

		while (i < digits.size()) {
			count = 0;

			while (i < digits.size() && digits[i] == 9) {
				count++;
				i++;
			}

			if (count > maxCount)
				maxCount = count;

			i++;
		}

		if (maxCount > divDigits * 4 / 5) {
			i = digits.size() - 1;

			do {
				count = 0;

				while (i > 0 && digits[i] != 9)
					i--;

				while (i > 0 && digits[i] == 9) {
					count++;
					i--;
				}
			} while (count != maxCount);

			digits.erase(digits.begin() + i + 1, digits.end());
			digits[i]++;
		}
	}
}

LongDoubleCPU::LongDoubleCPU() {
	sign = 1;
	digits = vector<int>(1, 0);
	exponent = 1;
}

LongDoubleCPU::LongDoubleCPU(const LongDoubleCPU& x) {
	sign = x.sign;
	exponent = x.exponent;
	digits = vector<int>(x.digits.size());

	for (size_t i = 0; i < x.digits.size(); i++)
		digits[i] = x.digits[i];
}

LongDoubleCPU::LongDoubleCPU(long double value) {
	stringstream ss;
	ss << setprecision(15) << value;

	initFromString(ss.str());
	removeZeroes();
}

LongDoubleCPU::LongDoubleCPU(const string& s) {
	initFromString(s);
	removeZeroes();
}

LongDoubleCPU& LongDoubleCPU::operator=(const LongDoubleCPU& x) {
	if (this == &x)
		return *this;

	sign = x.sign;
	exponent = x.exponent;
	digits = vector<int>(x.digits.size());

	for (size_t i = 0; i < x.digits.size(); i++)
		digits[i] = x.digits[i];

	return *this;
}

bool LongDoubleCPU::operator>(const LongDoubleCPU& x) const {
	if (sign != x.sign)
		return sign > x.sign;

	if (exponent != x.exponent)
		return (exponent > x.exponent) ^ (sign == -1);

	thrust::host_vector<int> d1(digits);
	thrust::host_vector<int> d2(x.digits);
	size_t size = max(d1.size(), d2.size());

	while (d1.size() != size)
		d1.push_back(0);

	while (d2.size() != size)
		d2.push_back(0);

	for (size_t i = 0; i < size; i++)
		if (d1[i] != d2[i])
			return (d1[i] > d2[i]) ^ (sign == -1);

	return false;
}

bool LongDoubleCPU::operator<(const LongDoubleCPU& x) const {
	return !(*this > x || *this == x);
}

bool LongDoubleCPU::operator>=(const LongDoubleCPU& x) const {
	return *this > x || *this == x;
}

bool LongDoubleCPU::operator<=(const LongDoubleCPU& x) const {
	return *this < x || *this == x;
}

bool LongDoubleCPU::operator==(const LongDoubleCPU& x) const {
	if (sign != x.sign)
		return false;

	if (exponent != x.exponent)
		return false;

	if (digits.size() != x.digits.size())
		return false;

	for (size_t i = 0; i < digits.size(); i++)
		if (digits[i] != x.digits[i])
			return false;

	return true;
}

bool LongDoubleCPU::operator!=(const LongDoubleCPU& x) const {
	return !(*this == x);
}

LongDoubleCPU LongDoubleCPU::operator-() const {
	LongDoubleCPU res(*this);
	res.sign = -sign;

	return res;
}

LongDoubleCPU LongDoubleCPU::operator+(const LongDoubleCPU& x) const {
	if (sign == x.sign) {
		long exp1 = exponent;
		long exp2 = x.exponent;
		long exp = max(exp1, exp2);

		thrust::host_vector<int> d1(digits);
		thrust::host_vector<int> d2(x.digits);

		while (exp1 != exp) {
			d1.insert(d1.begin(), 0);
			exp1++;
		}

		while (exp2 != exp) {
			d2.insert(d2.begin(), 0);
			exp2++;
		}

		size_t size = max(d1.size(), d2.size());

		while (d1.size() != size)
			d1.push_back(0);

		while (d2.size() != size)
			d2.push_back(0);

		size_t len = 1 + size;

		LongDoubleCPU res;

		res.sign = sign;
		res.digits = vector<int>(len, 0);

		for (size_t i = 0; i < size; i++)
			res.digits[i + 1] = d1[i] + d2[i];

		for (size_t i = len - 1; i > 0; i--) {
			res.digits[i - 1] += res.digits[i] / 10;
			res.digits[i] %= 10;
		}

		res.exponent = exp + 1;
		res.removeZeroes();

		return res;
	}

	if (sign == -1)
		return x - (-(*this));

	return *this - (-x);
}

LongDoubleCPU LongDoubleCPU::operator-(const LongDoubleCPU& x) const {
	if (sign == 1 && x.sign == 1) {
		bool cmp = *this > x;

		long exp1 = cmp ? exponent : x.exponent;
		long exp2 = cmp ? x.exponent : exponent;
		long exp = max(exp1, exp2);

		thrust::host_vector<int> d1(cmp ? digits : x.digits);
		thrust::host_vector<int> d2(cmp ? x.digits : digits);

		while (exp1 != exp) {
			d1.insert(d1.begin(), 0);
			exp1++;
		}

		while (exp2 != exp) {
			d2.insert(d2.begin(), 0);
			exp2++;
		}

		size_t size = max(d1.size(), d2.size());

		while (d1.size() != size)
			d1.push_back(0);

		while (d2.size() != size)
			d2.push_back(0);

		size_t len = 1 + size;

		LongDoubleCPU res;

		res.sign = cmp ? 1 : -1;
		res.digits = vector<int>(len, 0);

		for (size_t i = 0; i < size; i++)
			res.digits[i + 1] = d1[i] - d2[i];

		for (size_t i = len - 1; i > 0; i--) {
			if (res.digits[i] < 0) {
				res.digits[i] += 10;
				res.digits[i - 1]--;
			}
		}

		res.exponent = exp + 1;
		res.removeZeroes();

		return res;
	}

	if (sign == -1 && x.sign == -1)
		return (-x) - (-(*this));

	return *this + (-x);
}

LongDoubleCPU LongDoubleCPU::operator*(const LongDoubleCPU& x) const {
	size_t len = digits.size() + x.digits.size();

	LongDoubleCPU res;

	res.sign = sign * x.sign;
	res.digits = vector<int>(len, 0);
	res.exponent = exponent + x.exponent;

	for (size_t i = 0; i < digits.size(); i++)
		for (size_t j = 0; j < x.digits.size(); j++)
			res.digits[i + j + 1] += digits[i] * x.digits[j];

	for (size_t i = len - 1; i > 0; i--) {
		res.digits[i - 1] += res.digits[i] / 10;
		res.digits[i] %= 10;
	}

	res.removeZeroes();

	return res;
}

LongDoubleCPU LongDoubleCPU::operator/(const LongDoubleCPU& x) const {
	LongDoubleCPU res = *this * x.inverse();

	size_t intPart = max((long)0, exponent);

	if (intPart > res.digits.size() - 1)
		return res;

	size_t i = res.digits.size() - 1 - intPart;
	size_t n = max((long)0, res.exponent);

	if (i > n && res.digits[i] == 9) {
		while (i > n && res.digits[i] == 9)
			i--;

		if (res.digits[i] == 9) {
			res.digits.erase(res.digits.begin() + n, res.digits.end());
			res = res + res.sign;
		}
		else {
			res.digits.erase(res.digits.begin() + i + 1, res.digits.end());
			res.digits[i]++;
		}
	}

	return res;
}

LongDoubleCPU& LongDoubleCPU::operator+=(const LongDoubleCPU& x) {
	return (*this = *this + x);
}

LongDoubleCPU& LongDoubleCPU::operator-=(const LongDoubleCPU& x) {
	return (*this = *this - x);
}

LongDoubleCPU& LongDoubleCPU::operator*=(const LongDoubleCPU& x) {
	return (*this = *this * x);
}

LongDoubleCPU& LongDoubleCPU::operator/=(const LongDoubleCPU& x) {
	return (*this = *this / x);
}

LongDoubleCPU LongDoubleCPU::operator++(int) {
	LongDoubleCPU res(*this);
	*this = *this + 1;

	return res;
}

LongDoubleCPU LongDoubleCPU::operator--(int) {
	LongDoubleCPU res(*this);
	*this = *this - 1;

	return res;
}

LongDoubleCPU& LongDoubleCPU::operator++() {
	return (*this = *this + 1);
}

LongDoubleCPU& LongDoubleCPU::operator--() {
	return (*this = *this - 1);
}

LongDoubleCPU LongDoubleCPU::inverse() const {
	if (isZero())
		throw string("LongDoubleCPU LongDoubleCPU::inverse() - division by zero!");

	LongDoubleCPU x(*this);
	x.sign = 1;

	LongDoubleCPU d("1");

	LongDoubleCPU res;
	res.sign = sign;
	res.exponent = 1;
	res.digits = vector<int>();

	while (x < 1) {
		x.exponent++;
		res.exponent++;
	}

	while (d < x)
		d.exponent++;

	res.exponent -= d.exponent - 1;

	size_t numbers = 0;
	size_t intPart = max((long)0, res.exponent);
	size_t maxNumbers = divDigits + intPart;

	do {
		int div = 0;

		while (d >= x) {
			div++;
			d -= x;
		}

		d.exponent++;
		d.removeZeroes();

		res.digits.push_back(div);
		numbers++;
	} while (!d.isZero() && numbers < maxNumbers);

	return res;
}

LongDoubleCPU LongDoubleCPU::sqrt() const {
	if (sign == -1)
		throw string("LongDoubleCPU LongDoubleCPU::sqrt() - number is negative");

	if (isZero())
		return 0;

	LongDoubleCPU x0;
	LongDoubleCPU p("0.5");
	LongDoubleCPU xk("0.5");
	LongDoubleCPU eps;
	eps.digits = vector<int>(1, 1);
	eps.exponent = 1 - sqrtDigits;

	do {
		x0 = xk;
		xk = p * (x0 + *this / x0);
	} while ((x0 - xk).abs() > eps);

	xk.digits.erase(xk.digits.begin() + max((long)0, xk.exponent) + sqrtDigits, xk.digits.end());
	xk.removeZeroes();

	return xk;
}

LongDoubleCPU LongDoubleCPU::pow(const LongDoubleCPU& n) const {
	if (!n.isInteger())
		throw string("LongDoubleCPU LongDoubleCPU::power(const LongDoubleCPU& n) - n is not integer!");

	LongDoubleCPU res("1");
	LongDoubleCPU a = n.sign == 1 ? *this : this->inverse();
	LongDoubleCPU power = n.abs();

	while (power > 0) {
		if (power.isOdd())
			res *= a;

		a *= a;
		power /= 2;

		if (!power.isInteger())
			power.digits.erase(power.digits.end() - 1);
	}

	return res;
}

LongDoubleCPU LongDoubleCPU::abs() const {
	LongDoubleCPU res(*this);
	res.sign = 1;

	return res;
}

bool LongDoubleCPU::isInteger() const {
	if (exponent < 0)
		return false;

	return digits.size() <= (size_t)exponent;
}

bool LongDoubleCPU::isEven() const {
	if (!isInteger())
		return false;

	if (digits.size() == (size_t)exponent)
		return digits[digits.size() - 1] % 2 == 0;

	return true;
}

bool LongDoubleCPU::isOdd() const {
	if (!isInteger())
		return false;

	if (digits.size() == (size_t)exponent)
		return digits[digits.size() - 1] % 2 == 1;

	return false;
}

bool LongDoubleCPU::isZero() const {
	return digits.size() == 1 && digits[0] == 0;
}

int LongDoubleCPU::getSign() const
{
	return sign;
}

thrust::device_vector<int> LongDoubleCPU::getDigits() const
{
	 thrust::device_vector<int> diviceVector;
	 for (auto value : digits)
		 diviceVector.push_back(value);
	 return diviceVector;
}

long LongDoubleCPU::getExponent()const
{
	return exponent;
}

ostream& operator<<(ostream& os, const LongDoubleCPU& value) {
	if (value.sign == -1)
		os << '-';

	if (value.exponent > 0) {
		size_t i = 0;
		size_t e = value.exponent;

		while (i < value.digits.size() && i < e)
			os << value.digits[i++];

		while (i < e) {
			os << "0";
			i++;
		}

		if (i < value.digits.size()) {
			os << ".";

			while (i < value.digits.size())
				os << value.digits[i++];
		}
	}
	else if (value.exponent == 0) {
		os << "0.";

		for (size_t i = 0; i < value.digits.size(); i++)
			os << value.digits[i];
	}
	else {
		os << "0.";

		for (long i = 0; i < -value.exponent; i++)
			os << "0";

		for (size_t i = 0; i < value.digits.size(); i++)
			os << value.digits[i];
	}

	return os;
}



//GPU

void LongDoubleGPU::initFromString(const char* s, size_t length) {
	size_t index;

	if (s[0] == '-') {
		sign = -1;
		index = 1;
	}
	else {
		sign = 1;
		index = 0;
	}

	exponent = length - index;

	while (index < length) {
		if (s[index] == '.')
			exponent = sign == 1 ? index : index - 1;
		else
			digits.push_back(s[index] - '0');

		index++;
	}
}
void LongDoubleGPU::removeZeroes() {
	size_t n = max((long)1, exponent);

	while (digitsSize > n && digits[digitsSize - 1] == 0)
		digits.erase(digits.end() - 1);

	while (digitsSize> 1 && digits[0] == 0) {
		digits.erase(digits.begin());
		exponent--;
	}

	while (digitsSize > 1 && digits[digitsSize - 1] == 0)
		digits.erase(digits.end() - 1);

	if (isZero()) {
		exponent = 1;
		sign = 1;
	}

	normalize();
}

void LongDoubleGPU::normalize() {
	size_t start = max(exponent, (long)0);
	size_t realDigits = digits.size() - start;

	if (realDigits >= divDigits) {
		size_t count = 0;
		size_t maxCount = 0;

		size_t i = start;

		while (i < digits.size()) {
			count = 0;

			while (i < digits.size() && digits[i] == 9) {
				count++;
				i++;
			}

			if (count > maxCount)
				maxCount = count;

			i++;
		}

		if (maxCount > divDigits * 4 / 5) {
			i = digits.size() - 1;

			do {
				count = 0;

				while (i > 0 && digits[i] != 9)
					i--;

				while (i > 0 && digits[i] == 9) {
					count++;
					i--;
				}
			} while (count != maxCount);

			digits.erase(digits.begin() + i + 1, digits.end());
			digits[i]++;
		}
	}
}

LongDoubleGPU::LongDoubleGPU() {
	sign = 1;
	digits = thrust::device_vector<int>(1, 0);
	exponent = 1;
}

LongDoubleGPU::LongDoubleGPU(const LongDoubleGPU& x) {
	sign = x.sign;
	exponent = x.exponent;
	digits = thrust::device_vector<int>(x.digits.size());

	for (size_t i = 0; i < x.digits.size(); i++)
		digits[i] = x.digits[i];
}

LongDoubleGPU::LongDoubleGPU(const char* s, size_t length)
{
	initFromString(s, length);
	removeZeroes();
}

LongDoubleGPU::LongDoubleGPU(long double value) {
	stringstream ss; 
	ss << setprecision(15) << value;
	initFromString(ss.str().c_str(),ss.str().size());
	removeZeroes();
}

LongDoubleGPU::LongDoubleGPU(const LongDoubleCPU& x)
{
	sign = x.getSign();
	exponent = x.getExponent();
	
	digits = x.getDigits();
}


LongDoubleGPU& LongDoubleGPU::operator=(const LongDoubleGPU& x) {
	if (this == &x)
		return *this;

	sign = x.sign;
	exponent = x.exponent;
	digits = thrust::device_vector<int>(x.digits.size());

	for (size_t i = 0; i < x.digits.size(); i++)
		digits[i] = x.digits[i];

	return *this;
}

bool LongDoubleGPU::operator>(const LongDoubleGPU& x) const {
	if (sign != x.sign)
		return sign > x.sign;

	if (exponent != x.exponent)
		return (exponent > x.exponent) ^ (sign == -1);

	thrust::device_vector<int> d1(digits);
	thrust::device_vector<int> d2(x.digits);
	size_t size = max(d1.size(), d2.size());

	while (d1.size() != size)
		d1.push_back(0);

	while (d2.size() != size)
		d2.push_back(0);

	for (size_t i = 0; i < size; i++)
		if (d1[i] != d2[i])
			return (d1[i] > d2[i]) ^ (sign == -1);

	return false;
}

bool LongDoubleGPU::operator<(const LongDoubleGPU& x) const {
	return !(*this > x || *this == x);
}


bool LongDoubleGPU::operator==(const LongDoubleGPU& x) const {
	if (sign != x.sign)
		return false;

	if (exponent != x.exponent)
		return false;

	if (digits.size() != x.digits.size())
		return false;

	for (size_t i = 0; i < digits.size(); i++)
		if (digits[i] != x.digits[i])
			return false;

	return true;
}

bool LongDoubleGPU::operator!=(const LongDoubleGPU& x) const {
	return !(*this == x);
}

LongDoubleGPU LongDoubleGPU::operator-() const {
	LongDoubleGPU res(*this);
	res.sign = -sign;

	return res;
}

LongDoubleGPU LongDoubleGPU::operator+(const LongDoubleGPU& x) const {
	if (sign == x.sign) {
		long exp1 = exponent;
		long exp2 = x.exponent;
		long exp = max(exp1, exp2);

		thrust::device_vector<int> d1(digits);
		thrust::device_vector<int> d2(x.digits);

		while (exp1 != exp) {
			d1.insert(d1.begin(), 0);
			exp1++;
		}

		while (exp2 != exp) {
			d2.insert(d2.begin(), 0);
			exp2++;
		}

		size_t size = max(d1.size(), d2.size());

		while (d1.size() != size)
			d1.push_back(0);

		while (d2.size() != size)
			d2.push_back(0);

		size_t len = 1 + size;

		LongDoubleGPU res;

		res.sign = sign;
		res.digits = thrust::device_vector<int>(len, 0);

		for (size_t i = 0; i < size; i++)
			res.digits[i + 1] = d1[i] + d2[i];

		for (size_t i = len - 1; i > 0; i--) {
			res.digits[i - 1] += res.digits[i] / 10;
			res.digits[i] %= 10;
		}

		res.exponent = exp + 1;
		res.removeZeroes();

		return res;
	}

	if (sign == -1)
		return x - (-(*this));

	return *this - (-x);
}

LongDoubleGPU LongDoubleGPU::operator-(const LongDoubleGPU& x) const {
	if (sign == 1 && x.sign == 1) {
		bool cmp = *this > x;

		long exp1 = cmp ? exponent : x.exponent;
		long exp2 = cmp ? x.exponent : exponent;
		long exp = max(exp1, exp2);

		thrust::device_vector<int> d1(cmp ? digits : x.digits);
		thrust::device_vector<int> d2(cmp ? x.digits : digits);

		while (exp1 != exp) {
			d1.insert(d1.begin(), 0);
			exp1++;
		}

		while (exp2 != exp) {
			d2.insert(d2.begin(), 0);
			exp2++;
		}

		size_t size = max(d1.size(), d2.size());

		while (d1.size() != size)
			d1.push_back(0);

		while (d2.size() != size)
			d2.push_back(0);

		size_t len = 1 + size;

		LongDoubleGPU res;

		res.sign = cmp ? 1 : -1;
		res.digits = thrust::device_vector<int>(len, 0);

		for (size_t i = 0; i < size; i++)
			res.digits[i + 1] = d1[i] - d2[i];

		for (size_t i = len - 1; i > 0; i--) {
			if (res.digits[i] < 0) {
				res.digits[i] += 10;
				res.digits[i - 1]--;
			}
		}

		res.exponent = exp + 1;
		res.removeZeroes();

		return res;
	}

	if (sign == -1 && x.sign == -1)
		return (-x) - (-(*this));

	return *this + (-x);
}

LongDoubleGPU LongDoubleGPU::operator*(const LongDoubleGPU& x) const {
	size_t len = digits.size() + x.digits.size();

	LongDoubleGPU res;

	res.sign = sign * x.sign;
	res.digits = thrust::device_vector<int>(len, 0);
	res.exponent = exponent + x.exponent;

	for (size_t i = 0; i < digits.size(); i++)
		for (size_t j = 0; j < x.digits.size(); j++)
			res.digits[i + j + 1] += digits[i] * x.digits[j];

	for (size_t i = len - 1; i > 0; i--) {
		res.digits[i - 1] += res.digits[i] / 10;
		res.digits[i] %= 10;
	}

	res.removeZeroes();

	return res;
}

LongDoubleGPU LongDoubleGPU::operator/(const LongDoubleGPU& x) const {
	LongDoubleGPU res = *this * x.inverse();

	size_t intPart = max((long)0, exponent);

	if (intPart > res.digits.size() - 1)
		return res;

	size_t i = res.digits.size() - 1 - intPart;
	size_t n = max((long)0, res.exponent);

	if (i > n && res.digits[i] == 9) {
		while (i > n && res.digits[i] == 9)
			i--;

		if (res.digits[i] == 9) {
			res.digits.erase(res.digits.begin() + n, res.digits.end());
			res = res + res.sign;
		}
		else {
			res.digits.erase(res.digits.begin() + i + 1, res.digits.end());
			res.digits[i]++;
		}
	}

	return res;
}

LongDoubleGPU LongDoubleGPU::abs() const {
	LongDoubleGPU res(*this);
	res.sign = 1;

	return res;
}

__device__ LongDoubleGPU LongDoubleGPU::inverse() const
{
	
	LongDoubleGPU x(*this);
	x.sign = 1;

	LongDoubleGPU d("1",1);

	LongDoubleGPU res;
	res.sign = sign;
	res.exponent = 1;
	

	while (x < 1) {
		x.exponent++;
		res.exponent++;
	}

	while (d < x)
		d.exponent++;

	res.exponent -= d.exponent - 1;

	size_t numbers = 0;
	size_t intPart = max((long)0, res.exponent);
	size_t maxNumbers = divDigits + intPart;

	do {
		int div = 0;

		while (d > x||d==x) {
			div++;
			d =d- x;
		}

		d.exponent++;
		d.removeZeroes();

		res.digits.push_back(div);
		numbers++;
	} while (!d.isZero() && numbers < maxNumbers);

	return res;
}

bool LongDoubleGPU::isInteger() const {
	if (exponent < 0)
		return false;

	return digits.size() <= (size_t)exponent;
}

bool LongDoubleGPU::isEven() const {
	if (!isInteger())
		return false;

	if (digits.size() == (size_t)exponent)
		return digits[digits.size() - 1] % 2 == 0;

	return true;
}

bool LongDoubleGPU::isOdd() const {
	if (!isInteger())
		return false;

	if (digits.size() == (size_t)exponent)
		return digits[digits.size() - 1] % 2 == 1;

	return false;
}

bool LongDoubleGPU::isZero() const {
	return digits.size() == 1 && digits[0] == 0;
}

