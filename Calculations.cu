#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include "Calculations.cuh"
#include ""
#include <cstdio>
#define _USE_MATH_DEFINES
#include <math.h>
#include <iostream>

__device__ __constant__ uint8_t devColorPerIter[4*1549];

__global__ void MandelbrotSet( double numberPerPixel,
	 double leftTopX,  double leftTopY, uint8_t* result) {
	int startx = blockIdx.x * blockDim.x + threadIdx.x;
	int starty = blockIdx.y * blockDim.y + threadIdx.y;

	double x0 = (startx -WIDTH / 2 - leftTopX) * numberPerPixel;
	double y0 = (starty - HEIGHT / 2 - leftTopY) * numberPerPixel;
	 
	//Optimization,fast calculating big circle 
	double p = (x0 - 0.25) * (x0 - 0.25) + y0 * y0;
	double angle = atan2(y0, x0 - 0.25);
	double pc = 0.5 - (cos(angle) / 2);
	if (p <= pc * pc) {
		for (int r = 0; r <= 3; ++r) {	
			result[4 * (starty * WIDTH + startx) + r] = devColorPerIter[4 * iterations + r];
		}
		return;
	}

	double xPast = x0, yPast = y0;
	double R = 10;
	for (int i = 0; i < iterations; ++i) {
		 //Z^2
		 double	x = xPast * xPast - yPast * yPast + x0;
		 double	y =  2 * xPast * yPast + y0;
		if (x * x + y * y > R * R) {
			for (int r = 0; r <= 3; ++r)
				result[4 * (starty * WIDTH + startx) + r] = devColorPerIter[4 * i + r];
			return;
		}
		xPast = x;
		yPast = y;
	}

	for (int r = 0; r <= 3; ++r)
		result[4 * (starty * WIDTH + startx) + r] = devColorPerIter[4 * iterations + r]; 	
}

__global__ void JuliaSet(double numberPerPixel,
	double leftTopX, double leftTopY, uint8_t* result,double cx, double cy) {
	int startx = blockIdx.x * blockDim.x + threadIdx.x;
	int starty = blockIdx.y * blockDim.y + threadIdx.y;

	double x = (startx - WIDTH / 2 - leftTopX) * numberPerPixel;
	double y = (starty - HEIGHT / 2 - leftTopY) * numberPerPixel;
	double R = 36.;

	int pos = 4 * (starty * WIDTH + startx);
	for (int i = 0; i < iterations; ++i) {
		//Z^2
		double xCopy = x;
			x = xCopy * xCopy - y * y ;
			y = 2 * xCopy * y ;	
		//sin 
			xCopy = x;
			x = sin(xCopy) * cosh(y) + cx;
			y = cos(xCopy) * sinh(y) + cy;
		if(x*x+y*y>R){
			for(int r=0;r<=3;++r)
			result[pos + r] = devColorPerIter[4 * i + r];
			return;
		}
	}
	for (int r = 0; r <= 3; ++r)
		result[pos + r] = devColorPerIter[4 * iterations + r];
}

__global__ void parallelTransfer(int* pixelIteration,
	float deltaX, float deltaY) {
	int startx = blockIdx.x * blockDim.x + threadIdx.x;
	int starty = blockIdx.y * blockDim.y + threadIdx.y;

	int oldPosX = startx -deltaX;
	int oldPosY = startx - deltaY;
	if (oldPosX < 0 || oldPosY < 0 || oldPosX >= WIDTH || oldPosY >= HEIGHT)
		pixelIteration[starty*WIDTH+startx]=0;
	pixelIteration[starty * WIDTH + startx] = pixelIteration[oldPosY * WIDTH + oldPosX];

}

__host__
void render( double numberPerPixel,
	 double leftTopX,  double leftTopY, uint8_t* result,double deltaTime) {
	
	uint8_t* devResult = 0;
	
	if (hipMalloc((void**)&devResult, 4 * WIDTH * HEIGHT * sizeof(uint8_t)) != hipSuccess) {
		std::cerr << "Cuda malloc failed!";
		exit(EXIT_FAILURE);
	}
	dim3 threadsPerBlock(32, 16);
	dim3 numBlocks((WIDTH) / threadsPerBlock.x, (HEIGHT) / threadsPerBlock.y);

	//Calculating change x, y for Jukia Set 
	static double cx =0;
	static double cy=0;
	static double time=0;
	time += deltaTime;
	if (time > 5000) {
		cx += 0.01;
		cy += 0.01;
		time = 0;
		if (cx >= 2 || cy >= 2) {
			cx = 0;
			cy = 0;
		}
	}

	//MandelbrotSet << <numBlocks, threadsPerBlock >> > (numberPerPixel, leftTopX, leftTopY, 
		//devResult);
	JuliaSet << <numBlocks, threadsPerBlock >> > (numberPerPixel, leftTopX, leftTopY,
		devResult,cx,cy);

	hipError_t  error = hipGetLastError();
	if (error != hipSuccess)
	{
		std::cerr << "Error in set: "<<error;
		exit(EXIT_FAILURE);
	}
	if (hipMemcpy(result, devResult, 4 * WIDTH * HEIGHT * sizeof(uint8_t), hipMemcpyDeviceToHost) != hipSuccess)
	{
		std::cerr << "CudaMemcp Device to host failed!";
		exit(EXIT_FAILURE);
	}
	if (hipFree(devResult) != hipSuccess)
	{
		std::cerr << "CudaFree failed!";
		exit(EXIT_FAILURE);
	}
}

__host__ void defineColorPerIter(uint8_t* whitchColorPerIter)
{
	if (hipMemcpyToSymbol(HIP_SYMBOL(devColorPerIter), whitchColorPerIter, 4 * 1549 * sizeof(uint8_t)) != hipSuccess)
	{
		std::cerr << "CudaMemcpy failed!";
	}

}